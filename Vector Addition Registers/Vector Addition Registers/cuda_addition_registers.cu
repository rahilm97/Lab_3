#include "hip/hip_runtime.h"
//Vector Addition using CUDA.
//Winter 2020
//High Performance Computing.

#include <string>                                 //For stoi.
#include <iostream>                               //For stdout.
#include <cstdlib>                                //For random number generator.
#include <chrono>                                 //For getting time.
#include <climits>                                //For maximum n.
#include <cmath>

#include "hip/hip_runtime.h"                         //For Windows support.
#include ""

//The type that is used for the calculations.
typedef int type;

//Define constants for min/max.
#define RANDOMIZE_MIN -10
#define RANDOMIZE_MAX 10

//Cuda calculator which will run in each thread.
__global__ void cuda_calculator(type* a, type* b, type* c, int num_calcs)
{
    //Calculate the starting index.
    int start_index = (threadIdx.x + blockIdx.x * blockDim.x) * num_calcs;
    int end_index = start_index + num_calcs;

    //Add the vectors in the current thread index.
    #pragma unroll
    for (int i = start_index; i < end_index; i++)
        c[i] = a[i] + b[i];
}

//Cuda addition which runs the cuda program.
int cuda_addition(type* a, type* b, type* c, int n, int blocks,
    int threads, double times[3])
{
    //Create pointers for the GPU memory allocation
    type* cu_vec_a;
    type* cu_vec_b;
    type* cu_vec_c;

    //Calculate the number of elements that each kernel will handle (round up).
    int num_calcs = std::ceil((double)n / (((double)blocks) * ((double)threads)));

    //Calculate the padding (for output matrix to avoid conditionals in kernel.
    int padding_size = (int)(num_calcs * blocks * threads) - n;

    //Allocate memory on the device for the arrays.
    hipMalloc((void**)&cu_vec_a, sizeof(type) * (n + padding_size));
    hipMalloc((void**)&cu_vec_b, sizeof(type) * (n + padding_size));
    hipMalloc((void**)&cu_vec_c, sizeof(type) * (n + padding_size));

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    //Capture the beginning time before the data transfer (from host).
    auto begin_transfer_to = std::chrono::high_resolution_clock::now();

    //Copy the data, and the size from the main memory to VRAM.
    hipMemcpy(cu_vec_a, a, ((int)sizeof(type)) * n, hipMemcpyHostToDevice);
    hipMemcpy(cu_vec_b, b, ((int)sizeof(type)) * n, hipMemcpyHostToDevice);

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    //Calculate the total time in seconds that it took to transfer data to the device
    auto total_transfer_to = std::chrono::high_resolution_clock::now() - begin_transfer_to;
    times[0] = std::chrono::duration<double>(total_transfer_to).count();

    //Capture the beginning time before the calculations.
    auto begin_calcs_only = std::chrono::high_resolution_clock::now();

    //Launch the addition kernel on the device.
    cuda_calculator << <blocks, threads >> > (cu_vec_a, cu_vec_b, cu_vec_c, num_calcs);

    //Check if we got any errors.
    if (hipGetLastError() != hipSuccess)
        return EXIT_FAILURE;

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    //Calculate the total time in seconds that it took to calculate.
    auto total_calcs_only = std::chrono::high_resolution_clock::now() - begin_calcs_only;
    times[1] = std::chrono::duration<double>(total_calcs_only).count();

    //Capture the beginning time before the calculations.
    auto begin_transfer_from = std::chrono::high_resolution_clock::now();

    //Copy the results back from Vram to main ram.
    hipMemcpy(c, cu_vec_c, ((int)sizeof(type)) * n, hipMemcpyDeviceToHost);

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    //Calculate the total time in seconds that it took to transfer back to host.
    auto total_transfer_from = std::chrono::high_resolution_clock::now() - begin_transfer_from;
    times[2] = std::chrono::duration<double>(total_transfer_from).count();

    //Deallocate memory in the GPU.
    hipFree(cu_vec_a);
    hipFree(cu_vec_b);
    hipFree(cu_vec_c);

    //Wait for the thread to finish execution.
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}

//Sequential addition function.
double seq_addition(type* a, type* b, type* c, int size)
{
    //Capture the beginning time before the calculations.
    auto begin = std::chrono::high_resolution_clock::now();

    //Iterate over the vectors and add the elements.
    for (int i = 0; i < size; i++)
        c[i] = a[i] + b[i];

    //Calculate and return the total time in seconds that it took to compute.
    auto total = std::chrono::high_resolution_clock::now() - begin;
    return std::chrono::duration<double>(total).count();;
}

//Sequential subtraction function (used for residual matrix).
void seq_subtraction(type* a, type* b, type* c, int size)
{
    //Iterate over the vectors and subtract the elements.
    for (int i = 0; i < size; i++)
        c[i] = a[i] - b[i];
}

//Returns false if first and second aren't equal, true otherwise.
bool are_equal(type* first, type* second, int size)
{
    //Iterate over and return false if not equal.
    for (int i = 0; i < size; i++)
        if (first[i] != second[i])
            return false;

    //If we get here, they were equal.
    return true;
}

//A function which randomizes the vector, by defualt it only uses values between -10 - 10
void randomize(type* vec, int size, int min = RANDOMIZE_MIN, int max = RANDOMIZE_MAX)
{
    //Perform this to ensure the random number generation is truly random.
    std::srand(std::chrono::system_clock::now().time_since_epoch().count());

    //Iterate through, and generate random numbers for each index.
    for (int i = 0; i < size; i++)
        vec[i] = ((type)std::rand() %
        (type)(RANDOMIZE_MAX * 2) + (type)RANDOMIZE_MIN) % RANDOMIZE_MAX;
}

//Print the given vector to stdout.
void dump(type* vec, int size)
{
    //Iterate through, and generate random numbers for each index.
    for (int i = 0; i < size - 1; i++)
        std::cout << std::scientific << vec[i] << " | ";

    //Print the last item in a different format and add a new line.
    std::cout << std::scientific << vec[size - 1] << std::endl;
}

//A function which will be called when there is an error.
int error(std::string msg)
{
    //Print the error message.
    std::cout << "Error: " << msg << std::endl;

    //Print the usage message.
    std::cout << std::endl << "Usage Guide:" << std::endl
        << "\t* ./a.out <Size of Vectors> <Number of Blocks> <Number of Threads>"
        << " <Output Mode>" << std::endl << "\t* Output mode is either \'q\' "
        << "(quiet) or \'v\' (verbose)" << std::endl
        << "\t* Number of blocks and threads are for the GPU." << std::endl;

    //Return exit failure for passing it back to the terminal.
    return EXIT_FAILURE;
}

//Main method which parses the arguments, and runs the program.
int main(int argc, char** argv)
{
    //Define values for parameters.
    int n, blocks, threads;
    bool verbose;

    //Check for invalid number of args.
    if (argc != 5)
        return error("Invalid number of arguments.");

    //Parse the arguments.
    try
    {
        n = std::stoi(argv[1]);
        blocks = std::stoi(argv[2]);
        threads = std::stoi(argv[3]);
    }
    catch (...)      //If we get here, there was an error in the arguments.
    {
        return error("Invalid arguments, could not parse.");
    }

    //Check the print mode.
    if (std::string(argv[4]) == "q" || std::string(argv[4]) == "v")
        //If the mode is valid and set to v, set verbose to true, false otherwise.
        verbose = (std::string(argv[4]) == "v" ? true : false);
    else
        //If we get here an invalid mode was passed.
        return error("Invalid print mode.");

    //Check for invalid threads / blocks / n sizes.
    if (n < 1 || blocks < 1 || threads < 1)
        return error("Invalid arguments. All parameters should be positive.");

    //Check if we're gonna get overflow.
    if (n > INT_MAX)
        return error("Integer Overflow, please reduce N.");

    //Allocate memory for the input vectors.
    type* vec_a = new type[n];
    type* vec_b = new type[n];

    //Randomize the input vectors.
    randomize(vec_a, n);
    randomize(vec_b, n);

    //Allocate output matrices for the sequential and cuda executions.
    type* vec_c_seq = new type[n];
    type* vec_c_cuda = new type[n];

    //Perform the sequential addition.
    double seq_time = seq_addition(vec_a, vec_b, vec_c_seq, n);

    //Perform the cuda addition, and capture the timings.
    double times[3];
    int stat = cuda_addition(vec_a, vec_b, vec_c_cuda, n, blocks, threads, times);

    //Check the status.
    if (stat == EXIT_FAILURE)
        return error("Failed to execute kernel.");

    //Check if the cuda and sequential results are not equal (error).
    if (!are_equal(vec_c_seq, vec_c_cuda, n))
    {
        std::cout << "Error: Output vectors were not equal." << std::endl
            << "ErrorInfo: N=" << n << " Blocks=" << blocks
            << " Threads=" << threads << std::endl;
    }

    //Print the timing results, and the input arguments.
    std::cout << "[Cuda_Transfer_To_Device_Seconds]=" << std::scientific << times[0]
        << "  [Cuda_Transfer_To_Host_Seconds]=" << std::scientific << times[2]
        << "  [Cuda_Calculation_Time_Seconds]=" << std::scientific << times[1]
        << "  [Sequential_Time_Seconds]=" << std::scientific << seq_time
        << "  [N]=" << n << "  [Blocks]=" << blocks
        << "  [Threads]=" << threads
        << std::endl;

    //Allocate memory for residual vector.
    type* residual = new type[n];

    //Check if we're in verbose output mode.
    if (verbose)
    {
        //Calculate residual vector for sequential implementation vs cuda.
        seq_subtraction(vec_c_seq, vec_c_cuda, residual, n);

        //Print out the inputs, calculations and residual vector.
        std::cout << std::endl << "Printing out the First Vector:" << std::endl;
        dump(vec_a, n);

        std::cout << "\nPrinting out the Second Vector:" << std::endl;
        dump(vec_b, n);

        std::cout << "\nPrinting out the Addition results (Sequential):" << std::endl;
        dump(vec_c_seq, n);

        std::cout << "\nPrinting out the Addition results (Cuda):" << std::endl;
        dump(vec_c_cuda, n);

        std::cout << "\nPrinting out the residual matrix (Seq - Cuda):" << std::endl;
        dump(residual, n);
    }

    //Deallocate the memory in the heap.
    delete[] vec_a, vec_b, vec_c_seq, vec_c_cuda, residual;

    return EXIT_SUCCESS;
}
